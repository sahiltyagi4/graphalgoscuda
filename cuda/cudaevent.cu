
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cudaEventLoop(int N){
    for (int i=0; i<N; i++) {
        printf("index %d\n", i);
    }
}

int main(){
    int N = 100;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cudaEventLoop<<<4,128>>>(N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time %f milliseconds\n", milliseconds);
}
