#include "hip/hip_runtime.h"
#include <stdio.h>

//nvcc -arch=sm_70 -o test-gpu testgpu.cu -run
void runCPU() {
    printf("executing on CPU.\n")
}

__global__ void runGPU() {
    if(threadIdx.x == 127 && blockIdx.x == 3) {
        printf('executing on GPU.\n')
    }
}

int main() {
    runCPU();
    runGPU<<<4,128>>>();
    hipDeviceSynchronize();
}